#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#ifdef __cplusplus
extern "C" {
#endif /*__cplusplus*/
#include <stdio.h> // stdio functions are used since C++ streams aren't necessarily thread safe
#include <stdlib.h>
#include <string.h>
#include "graphio.h"
#include "graph.h"
#ifdef __cplusplus
}
#endif /*__cplusplus*/

#include <string>

//#include <omp.h>

//#define DEBUG
#define NREPS 10 // number of repetations for time calculations

#define THREADS_PER_BLOCK 1024

__global__ void ClosenessCentKernel(int *result, const etype *rowPtr, const vtype *colInd, vtype nov)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (index < nov)
		result[index] = -1;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CudaClosenessCent(int *result, const etype *rowPtr, const vtype *colInd, vtype nov)
{
	etype *dev_rowPtr = 0;
	vtype *dev_colInd = 0;
	int *dev_result = 0;
	hipError_t cudaStatus;

	//===========================================================================================================================
	// Allocate GPU buffers for three vectors (two input, one output)
	cudaStatus = hipMalloc((void**)&dev_result, nov * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_rowPtr, nov * sizeof(etype));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_colInd, rowPtr[nov] * sizeof(vtype));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//===========================================================================================================================
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_rowPtr, rowPtr, nov * sizeof(etype), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_colInd, colInd, rowPtr[nov] * sizeof(vtype), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//===========================================================================================================================
	// Launch a kernel on the GPU with one thread for each element, and check for errors.
	int numThreads = (int)sqrt(THREADS_PER_BLOCK);
	dim3 dimBlock(numThreads, numThreads, 1);
	//dim3 dimGrid(nov / numThreads, nov / numThreads);
	printf("%d, %d\n", nov, nov / numThreads);
	ClosenessCentKernel<<<(nov+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, dimBlock>>>(dev_result, dev_rowPtr, dev_colInd, nov);

	//===========================================================================================================================
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, nov * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_rowPtr);
	hipFree(dev_colInd);

	return cudaStatus;
}
