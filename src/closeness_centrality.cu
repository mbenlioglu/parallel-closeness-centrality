#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#ifdef __cplusplus
extern "C" {
#endif /*__cplusplus*/
#include <stdio.h> // stdio functions are used since C++ streams aren't necessarily thread safe
#include <stdlib.h>
#include <string.h>
#include "graphio.h"
#include "graph.h"
#ifdef __cplusplus
}
#endif /*__cplusplus*/

#include <string>

//#include <omp.h>

//#define DEBUG
#define NREPS 10 // number of repetations for time calculations

#define THREADS_PER_BLOCK 1024

__global__ void ClosenessCentKernel(int *result, const etype *rowPtr, const vtype *colInd, vtype nov)
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	unsigned int index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (index < nov)
		result[index] = -1;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t CudaClosenessCent(int *result, const etype *rowPtr, const vtype *colInd, vtype nov)
{
	etype *dev_rowPtr = 0;
	vtype *dev_colInd = 0;
	int *dev_result = 0;
	hipError_t cudaStatus;

	//===========================================================================================================================
	// Allocate GPU buffers for three vectors (two input, one output)
	cudaStatus = hipMalloc((void**)&dev_result, nov * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_rowPtr, nov * sizeof(etype));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_colInd, rowPtr[nov] * sizeof(vtype));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//===========================================================================================================================
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_rowPtr, rowPtr, nov * sizeof(etype), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_colInd, colInd, rowPtr[nov] * sizeof(vtype), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//===========================================================================================================================
	// Launch a kernel on the GPU with one thread for each element, and check for errors.
	int numThreads = (int)sqrt(THREADS_PER_BLOCK);
	dim3 dimBlock(numThreads, numThreads, 1);
	//dim3 dimGrid(nov / numThreads, nov / numThreads);
	printf("%d, %d\n", nov, nov / numThreads);
	ClosenessCentKernel<<<(nov+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, dimBlock>>>(dev_result, dev_rowPtr, dev_colInd, nov);

	//===========================================================================================================================
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, nov * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_rowPtr);
	hipFree(dev_colInd);

	return cudaStatus;
}

void PrintUsage(const char *appName)
{
	printf("./%s <graph_location> <GPU deviceID0\n>", appName);
}

int main(int argc, char *argv[])
{
	std::string baseName = std::string(argv[0]);
	std::string fillerAsterisk(100, '*');
	std::string fillerDashes(100, '-');

	// Get executable name from path
#ifdef WIN32
	baseName = baseName.substr(baseName.rfind('\\') + 1);
#else
	baseName = baseName.substr(baseName.rfind('/') + 1);
#endif // WIN32

	printf((fillerAsterisk + "\n").c_str());
	printf("Starting %s ...\n", baseName.c_str());
	printf((fillerAsterisk + "\n").c_str());

	printf("\nInitializing Device...\n");
	
	//===========================================================================================================================
	// set the CUDA capable GPU to be used
	//
	int num_gpus = 0;   // number of CUDA GPUs
	int devID = atoi(argv[2]); // selected device id
	
	hipGetDeviceCount(&num_gpus);

	if (num_gpus < 1)
	{
		printf("no CUDA capable devices were detected\n");
		return EXIT_FAILURE;
	}
	else if (devID > num_gpus || devID < 0)
	{
		printf("Invalid device ID\n");
		return EXIT_FAILURE;
	}

	hipDeviceProp_t dprop;
	hipError_t cudaStatus = hipGetDeviceProperties(&dprop, devID);
	printf("   %s #%d: %s\n\n","Selected Device", devID, dprop.name);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	hipSetDevice(devID);

	//===========================================================================================================================
	// read Graph
	//
	etype *rowPtr;
	vtype *colInd;
	ewtype *ewghts;
	vwtype *vwghts;
	vtype nov;

	printf("Reading graph... ");
	// Graph reading
	if (argc < 3)
	{
		PrintUsage(baseName.c_str());
		return EXIT_FAILURE;
	}

	if (read_graph(argv[1], &rowPtr, &colInd, &ewghts, &vwghts, &nov, 0) == -1)
	{
		printf("error in graph read\n");
		return EXIT_FAILURE;
	}
	printf("done.\n");
	printf((fillerDashes + "\n").c_str());
	//===========================================================================================================================

	int *result = new int[nov]();

    // Calculate closeness-centrality in parallel.
    cudaStatus = CudaClosenessCent(result, rowPtr, colInd, nov);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CudaClosenessCent failed!");
        return EXIT_FAILURE;
    }

	for (size_t i = 0; i < nov; i++)
	{
		if (result[i] != -1)
		{
			printf("WRONG! %d=%d\n", i, result[i]);
			break;
		}
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
